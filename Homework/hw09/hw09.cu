#include "hip/hip_runtime.h"
// Optimized using shared memory and on chip memory
// nvcc hw09.cu -o hw09 -lglut -lm -lGLU -lGL
//To stop hit "control c" in the window you launched it from.
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 32768
#define halfN	N/2
#define BLOCK 256

#define XWindowSize 2500
#define YWindowSize 2500

#define DRAW 10
#define DAMP 1.0

#define DT 0.001
#define STOP_TIME 10.0

#define G 1.0
#define H 1.0

#define EYE 45.0
#define FAR 90.0

// Globals
float4 p[N];
float3 v[N], f[N];
float4 *pos;
float3 *vel, *force;
FILE *data_file, *data_file1, *data_file2;
dim3 block, grid;
int NumberOfGpus, Gpu0Access, Gpu1Access;
const bool UseMultipleGPU = 1;

void CUDAerrorCheck(const char *message){
  hipError_t  error;
  error = hipGetLastError();

  if(error != hipSuccess){
    printf("\n CUDA ERROR: %s = %s\n", message, hipGetErrorString(error));
    exit(0);
  }
}

void set_initail_conditions(){
	int i,j,k,num,particles_per_side;
	float position_start, temp;
	float initial_seperation;

	temp = pow((float)N,1.0/3.0) + 0.99999;
	particles_per_side = temp;
	printf("\n cube root of N = %d \n", particles_per_side);
	position_start = -(particles_per_side -1.0)/2.0;
	initial_seperation = 2.0;

	for(i=0; i<N; i++) p[i].w = 1.0;

	num = 0;
	for(i=0; i<particles_per_side; i++){
		for(j=0; j<particles_per_side; j++){
			for(k=0; k<particles_per_side; k++){
				if(N <= num) break;
				p[num].x = position_start + i*initial_seperation;
				p[num].y = position_start + j*initial_seperation;
				p[num].z = position_start + k*initial_seperation;
				v[num].x = 0.0;
				v[num].y = 0.0;
				v[num].z = 0.0;
				num++;
			}
		}
	}

	block.x = BLOCK;
	block.y = 1;
	block.z = 1;

	grid.x = (N-1)/block.x + 1;
	grid.y = 1;
	grid.z = 1;

	hipMalloc( (void**)&pos, N *sizeof(float4) );
	hipMalloc( (void**)&vel, N *sizeof(float3) );
	hipMalloc( (void**)&force, N *sizeof(float3) );
}

void draw_picture(){
	int i;

	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);

	glColor3d(1.0,1.0,0.5);
	for(i=0; i<N; i++){
		glPushMatrix();
		glTranslatef(p[i].x, p[i].y, p[i].z);
		glutSolidSphere(0.1,20,20);
		glPopMatrix();
	}

	glutSwapBuffers();
}

__device__ float3 getBodyBodyForce(float4 p0, float4 p1){
	float3 f;
	float dx = p1.x - p0.x;
	float dy = p1.y - p0.y;
	float dz = p1.z - p0.z;
	float r2 = dx*dx + dy*dy + dz*dz;
	float r = sqrt(r2);

	float force  = (G*p0.w*p1.w)/(r2) - (H*p0.w*p1.w)/(r2*r2);

	f.x = force*dx/r;
	f.y = force*dy/r;
	f.z = force*dz/r;

	return(f);
}

__global__ void getForcesCollisionDoubleGPU(float4 *pos, float3 *vel, float3 *force){
	int j,ii;
	float3 force_mag, forceSum;
	float4 posMe;
	__shared__ float4 shPos[BLOCK];
	int id = threadIdx.x + blockDim.x*blockIdx.x;

	forceSum.x = 0.0;
	forceSum.y = 0.0;
	forceSum.z = 0.0;

	posMe.x = pos[id].x;
	posMe.y = pos[id].y;
	posMe.z = pos[id].z;
	posMe.w = pos[id].w;

	for(j=0; j<gridDim.x; j++){
		shPos[threadIdx.x] = pos[threadIdx.x + blockDim.x*j];
		__syncthreads();

		#pragma unroll 32
		for(int i=0; i<blockDim.x; i++){
			ii = i + blockDim.x*j;
			if(ii != id && ii < N){
				force_mag = getBodyBodyForce(posMe, shPos[i]);
				forceSum.x += force_mag.x;
				forceSum.y += force_mag.y;
				forceSum.z += force_mag.z;
			}
		}
	}

	if(id < N){
		force[id].x = forceSum.x;
		force[id].y = forceSum.y;
		force[id].z = forceSum.z;
	}
}

__global__ void moveBodiesCollisionDoubleGPU(float4 *pos, float3 *vel, float3 *force){
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	if(id<N){
		vel[id].x += ((force[id].x-DAMP*vel[id].x)/pos[id].w)*DT;
		vel[id].y += ((force[id].y-DAMP*vel[id].y)/pos[id].w)*DT;
		vel[id].z += ((force[id].z-DAMP*vel[id].z)/pos[id].w)*DT;

		pos[id].x += vel[id].x*DT;
		pos[id].y += vel[id].y*DT;
		pos[id].z += vel[id].z*DT;
	}
}

void getNumberOfGPUs(){
	hipGetDeviceCount(&NumberOfGpus);
	printf("\n***** You have %d GPUs available\n", NumberOfGpus);
}

void checkPeerToPeerAccess(){
	if(1 < NumberOfGpus && UseMultipleGPU == 1){
		hipDeviceCanAccessPeer(&Gpu0Access,0,1);
		hipDeviceCanAccessPeer(&Gpu1Access,1,0);

		printf("\n***** You will be using %d GPUs\n", NumberOfGpus);

		if(Gpu0Access == 0) printf("\nTSU Error: Device0 can not do peer to peer\n");
		if(Gpu1Access == 0) printf("\nTSU Error: Device1 can not do peer to peer\n");

		hipDeviceEnablePeerAccess(1,0);
	}
}

void n_body(){
	float dt;
	int   tdraw = 0;
	float time = 0.0;
	float elapsedTime;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	dt = DT;

    float4 *pos_gpu0, *pos_gpu1, *pos0, *pos1;
    float3 *vel_gpu0, *vel_gpu1, *vel0, *vel1;

    pos_gpu0 = pos;
    pos_gpu1 = pos+N/2;
    vel_gpu0 = vel;
    vel_gpu1 = vel+N/2;

	hipSetDevice(0);
	hipMemcpy(pos_gpu0, pos0, N/2*sizeof(float4), hipMemcpyHostToDevice);
	//CUDAerrorCheck("gpu position copy...\n");
	hipMemcpy(vel_gpu0, vel0, N/2*sizeof(float3), hipMemcpyHostToDevice);
	//CUDAerrorCheck("gpu velocity copy...\n");

	hipSetDevice(1);
	hipMemcpy(pos_gpu1, pos1+N/2, (N-N/2)*sizeof(float4), hipMemcpyHostToDevice);
	//CUDAerrorCheck("gpu position copy...\n");
	hipMemcpy(vel_gpu1, vel1+N/2, (N-N/2)*sizeof(float3), hipMemcpyHostToDevice);
	//CUDAerrorCheck("gpu velocity copy...\n");

	block.x = BLOCK;
	block.y = 1;
	block.z = 1;

	grid.x = (N-1)/block.x + 1;
	grid.y = 1;
	grid.z = 1;

	while(time < STOP_TIME){
		hipSetDevice(0);
		getForcesCollisionDoubleGPU<<<grid, block>>>(pos_0, vel_0, force);
		//CUDAerrorCheck("gpu0 force kernel...\n");
		moveBodiesCollisionDoubleGPU<<<grid, block>>>(pos_0, vel_0, force);
		//CUDAerrorCheck("gpu0 move kernel...\n");

		hipSetDevice(1);
		getForcesCollisionDoubleGPU<<<grid, block>>>(pos_1, vel_1, force);
		//CUDAerrorCheck("gpu1 force kernel...\n");
		moveBodiesCollisionDoubleGPU<<<grid, block>>>(pos_1, vel_1, force);
		//CUDAerrorCheck("gpu1 move kernel...\n");

		hipDeviceSynchronize();

		hipSetDevice(0);
		hipMemcpyPeerAsync(pos_1,1,pos_0,0,(N/2)*sizeof(float4));
		hipMemcpyPeerAsync(vel_1,1,vel_0,0,(N/2)*sizeof(float4));

		hipDeviceSynchronize();

		//To kill the draw comment out the next 7 lines.
		/*if(tdraw == DRAW) {
			hipMemcpy(p, p_GPU, N *sizeof(float4), hipMemcpyDeviceToHost);
			draw_picture();
			tdraw = 0;
		}
		tdraw++;*/

		time += dt;
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("\n\nGPU time = %3.1f milliseconds\n", elapsedTime);
	//hipMemcpy( p, p_GPU, N *sizeof(float4), hipMemcpyDeviceToHost );
}

void control(){
	set_initail_conditions();
	draw_picture();
	n_body();
	draw_picture();

	printf("\n DONE \n");
	while(1);
}

void Display(void){
	gluLookAt(EYE, EYE, EYE, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	control();
}

void reshape(int w, int h){
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, 0.2, FAR);
	glMatrixMode(GL_MODELVIEW);
}

int main(int argc, char** argv){
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("2 Body 3D");
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutMainLoop();
	return 0;
}
